#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdio>
#include <numeric>
#include <cmath>

#include "load.hpp"
#include "common.cuh"
#include "spmv/spmv.h"

using namespace std;

using index_t = int;
using offset_t = int;
using value_t = float;

constexpr int TEST_TIMES = 20;

int main(int argc, char** argv) {
    if (argc < 3) {
        std::cerr << "usage: ./bin/<program-name>  <filename.mtx>  <SpMV_kind_string>..." << std::endl;
        exit(1);        
    }

    csr_t<index_t, offset_t, value_t> csr =
        ToCsr(LoadCoo<index_t, offset_t, value_t>(argv[1]));

    vector<string> spmv_kind_strs(argv + 2, argv + argc);

    index_t n_rows, n_cols;
    offset_t nnz;
    n_rows = csr.number_of_rows;
    n_cols = csr.number_of_columns;
    nnz = csr.number_of_nonzeros;

    vector<value_t> vec_x(n_cols, 1);

    vector<value_t> vec_y(n_rows, value_t(0));

    //--------------------------------------------------------------------------
    // Device memory management

    index_t *dA_csrOffsets, *dA_columns;
    value_t *dA_values;
    value_t *dX, *dY;

    CHECK_CUDA(
        hipMalloc((void **)&dA_csrOffsets, (n_rows + 1) * sizeof(index_t)));
    CHECK_CUDA(hipMalloc((void **)&dA_columns, nnz * sizeof(index_t)));
    CHECK_CUDA(hipMalloc((void **)&dA_values, nnz * sizeof(value_t)));

    CHECK_CUDA(hipMalloc((void **)&dX, n_cols * sizeof(value_t)));
    CHECK_CUDA(hipMalloc((void **)&dY, n_rows * sizeof(value_t)));

    CHECK_CUDA(hipMemcpy(dA_csrOffsets, csr.row_offsets.data(),
                          (n_rows + 1) * sizeof(index_t),
                          hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dA_columns, csr.column_indices.data(), nnz * sizeof(index_t),
                          hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dA_values, csr.nonzero_values.data(), nnz * sizeof(value_t),
                          hipMemcpyHostToDevice));

    CHECK_CUDA(
        hipMemcpy(dX, vec_x.data(), n_cols * sizeof(value_t), hipMemcpyHostToDevice));
    CHECK_CUDA(
        hipMemcpy(dY, vec_y.data(), n_rows * sizeof(value_t), hipMemcpyHostToDevice));

    //--------------------------------------------------------------------------
    // CPU SpMV baseline
    vector<value_t> correct_y(n_rows, 0);
    SpMV_cpu_navie(n_rows, n_cols, nnz, csr.row_offsets.data(),
                   csr.column_indices.data(), csr.nonzero_values.data(),
                   vec_x.data(), correct_y.data());

    printf("Compute delta:\n");
    for (const auto& kind : spmv_kind_strs) {
        //--------------------------------------------------------------------------
        // SpMV APIs
        SpMV(kind, n_rows, n_cols, nnz, dA_csrOffsets, dA_columns, dA_values, dX, dY);
        CHECK_CUDA(hipMemcpy(vec_y.data(), dY, n_rows * sizeof(value_t), hipMemcpyDeviceToHost));

        //--------------------------------------------------------------------------
        // device results check
        double delta = 0.;
        for (int i = 0; i < n_rows; ++i) {
            delta += abs(correct_y[i] - vec_y[i]);
        }
        printf("[%-12s] sum: %12lf  avg: %12lf\n", kind.data(), delta, delta / n_rows);
    }
    printf("\n");


    printf("Time cost:\n");
    for (const auto& kind: spmv_kind_strs) {
        //--------------------------------------------------------------------------
        // time cost
        int64_t total_time = 0, kernel_time = 0;
        for (int i = 0; i < TEST_TIMES; ++i) {
            SpMV(kind, n_rows, n_cols, nnz, dA_csrOffsets, dA_columns, dA_values, dX, dY);
            total_time += Timer::total_cost();
            kernel_time += Timer::kernel_cost();
        }
        printf("[%-12s] total: %12lfms  kernel: %12lfms\n", 
            kind.data(), 1. * total_time / TEST_TIMES, 1. * kernel_time / TEST_TIMES);
    }

    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA(hipFree(dA_csrOffsets))
    CHECK_CUDA(hipFree(dA_columns))
    CHECK_CUDA(hipFree(dA_values))
    CHECK_CUDA(hipFree(dX))
    CHECK_CUDA(hipFree(dY))

    return EXIT_SUCCESS;
}
